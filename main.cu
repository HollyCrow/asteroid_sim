#include "hip/hip_runtime.h"
// /opt/cuda/bin/nvcc main.cu -o "GE" -diag-suppress 177 -diag-suppress 549 -ccbin=/usr/bin/clang -lSDL2 && time ./GE
#include <iostream>
#include <SDL2/SDL.h>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <thread>

#define width 1000
#define height 1000
#define G (double (6.6743e-11))

#define NUM_ASTEROIDS_X 1024 // Number of blocks
#define NUM_ASTEROIDS_Y 1024  // Block size
#define NUM_ASTEROIDS NUM_ASTEROIDS_X*NUM_ASTEROIDS_Y // Parameters for asteroid gen
#define MAX_XY 1e12
#define MAX_VELOCITY 1e4

SDL_Window* window = nullptr;
SDL_Renderer* renderer = nullptr;
bool running = true;
bool paused = false;
long scale = 2e9;
int time_step = 1000;
long offset_x = 490;
long offset_y = 490;
double big_bodies[2*5] = {0, 0, 0, 0, 1.989e30, 7.78e11, 0, 0, 1.306e4, 1.898e27}; // Mass e-23, distance e-6.
double big_bodies_G_shite[2] = {};
double small_bodies[NUM_ASTEROIDS*4];
double * cuda_small_bodies;
double * cuda_big_bodies;
double * cuda_big_bodies_G_shite;

void draw_big_bodies(){
    SDL_Rect body_rect{0, 0, 10, 10};
    SDL_SetRenderDrawColor(renderer, 0,255,255,255);
    for (long i = 0; i < 2; i++){
        body_rect.x = int(long(big_bodies[i*5]) / scale + offset_x);
        body_rect.y = int(long(big_bodies[i*5+1]) / scale + offset_y);
        SDL_RenderFillRect(renderer, &body_rect);
    }
}
void draw_small_bodies(){
    SDL_SetRenderDrawColor(renderer, 255,255,255,255);
    for (long i = 0; i < NUM_ASTEROIDS; i++){
        SDL_RenderDrawPoint(renderer, int(long(small_bodies[i*4+0])/scale + offset_x), int(long(small_bodies[i*4+1])/scale + offset_y));
    }
}
void draw(){
    SDL_SetRenderDrawColor(renderer, 0,0,0,255);
    SDL_RenderClear(renderer);
    draw_big_bodies();
    draw_small_bodies();
    SDL_RenderPresent(renderer);
};

double mag(double x, double y){
    return sqrt((x*x)+(y*y));
}

void big_phys(){
    for (long i = 0; i < 2; i++)for (long j = 0; j < 2; j++){
            if (i == j) { continue;}
            double r = mag(big_bodies[i*5+0]-big_bodies[j*5+0], big_bodies[i*5+1]-big_bodies[j*5+1]);
            double acc_mag_over_r = (-G * big_bodies[j*5+4] ) / (r*r*r);
            big_bodies[i*5+2]+= acc_mag_over_r*(big_bodies[i*5+0]-big_bodies[j*5+0])*time_step;
            big_bodies[i*5+3]+= acc_mag_over_r*(big_bodies[i*5+1]-big_bodies[j*5+1])*time_step;
    };

    for (long i = 0; i < 2; i++){
        big_bodies[i*5+0]+= big_bodies[i*5+2]*time_step;
        big_bodies[i*5+1]+= big_bodies[i*5+3]*time_step;
    }
}
void randomise_asteroids() {
    std::srand(static_cast<unsigned>(std::time(0)));
    for (long i = 0; i < NUM_ASTEROIDS; ++i) {
        // Randomize position
        double angle = (std::rand() / (double)RAND_MAX) * 2 * M_PI;
        double distance = ((std::rand() / (double)RAND_MAX)*0.5 + 0.5) * MAX_XY; // Ensure non-zero distance
        small_bodies[i * 4 + 0] = distance * std::cos(angle); // x
        small_bodies[i * 4 + 1] = distance * std::sin(angle); // y

        // Compute velocity for circular orbit
        double speed = MAX_VELOCITY * (distance / MAX_XY);
        small_bodies[i * 4 + 2] = -speed * std::sin(angle); // vx (perpendicular)
        small_bodies[i * 4 + 3] = speed * std::cos(angle);  // vy (perpendicular)
    }
}

__device__ double mag_dev(double x, double y){
    return sqrt((x*x)+(y*y));
}

__global__ void asteriod_phys(double * small_bodies, double * big_bodies, double * big_bodies_G_shite, int time_step){
    long idx = threadIdx.x+blockDim.x*blockIdx.x;
    if (idx >= NUM_ASTEROIDS)return;

    for (long i = 0; i < 2; i++){
        double r_x = small_bodies[idx*4+0]-big_bodies[i*5+0];
        double r_y = small_bodies[idx*4+1]-big_bodies[i*5+1];
        double r = mag_dev(r_x, r_y);
        if (r < (i == 0)? 5e8 : 1e7){

        }

        double acc_mag_over_r = big_bodies_G_shite[i] / (r*r*r); // -G * big_bodies[n] is recomputed for every asteroid. TODO: Fix that.
        small_bodies[idx*4+2]+= acc_mag_over_r*(r_x)*time_step;
        small_bodies[idx*4+3]+= acc_mag_over_r*(r_y)*time_step;
    };
    small_bodies[idx*4+0] += small_bodies[idx*4+2]*time_step;
    small_bodies[idx*4+1] += small_bodies[idx*4+3]*time_step;
}

void small_phys(){
    if (paused) return;
    hipMalloc(&cuda_big_bodies, sizeof (big_bodies));
    hipMemcpy(cuda_big_bodies, &big_bodies, sizeof (big_bodies), hipMemcpyHostToDevice);
    for (int i = 0; i < 2; i++){
        big_bodies_G_shite[i] = (-G * big_bodies[i*5+4] );
    }
    hipMalloc(&cuda_big_bodies_G_shite, sizeof (big_bodies_G_shite));
    hipMemcpy(cuda_big_bodies_G_shite, &big_bodies_G_shite, sizeof (big_bodies_G_shite), hipMemcpyHostToDevice);

    asteriod_phys<<<NUM_ASTEROIDS_X, NUM_ASTEROIDS_Y>>>(cuda_small_bodies, cuda_big_bodies, big_bodies_G_shite, time_step);
    hipDeviceSynchronize();
}

void fixed_update(){
    while (running) {
        big_phys();
        small_phys();
    }
}

int main() {
    randomise_asteroids();

    hipMalloc(&cuda_small_bodies, sizeof (small_bodies));
    hipMemcpy(cuda_small_bodies, &small_bodies, sizeof (small_bodies), hipMemcpyHostToDevice);
    hipMalloc(&cuda_big_bodies, sizeof (big_bodies));
    hipMemcpy(cuda_big_bodies, &big_bodies, sizeof (big_bodies), hipMemcpyHostToDevice);

//    srand((unsigned) time(NULL));
    std::cout << "Asteroid sim... (Can't spell it :3)" << std::endl;

    SDL_Init(SDL_INIT_VIDEO);
    SDL_CreateWindowAndRenderer(width, height, 0, &window, &renderer);
    SDL_RenderSetScale(renderer,1,1);
    SDL_SetWindowTitle(window, "Asteroid Sim");


    std::thread physicsThread(&fixed_update);
    int a,b=0;


    while (running) {
        a = SDL_GetTicks();
        if (a-b > 1000/60){
            hipMemcpy(small_bodies, cuda_small_bodies, sizeof(small_bodies), hipMemcpyDeviceToHost);
            draw();
            b=a;
        }
        SDL_Event event;
        while (SDL_PollEvent(&event)) {
            switch (event.type) {
                case SDL_QUIT:
                    running = false;
                    break;
                case SDL_KEYDOWN:
                    if (event.key.keysym.sym ==SDLK_SPACE)paused = !paused;
                    if (event.key.keysym.sym ==SDLK_DOWN){time_step*=0.1;};
                    if (event.key.keysym.sym ==SDLK_UP){time_step*=10;};
                    if (event.key.keysym.sym ==SDLK_LEFT){scale*=0.5;};
                    if (event.key.keysym.sym ==SDLK_RIGHT){scale*=2;};
                    if (event.key.keysym.sym ==SDLK_w){offset_y-=100;}
                    if (event.key.keysym.sym ==SDLK_s){offset_y+=100;}
                    if (event.key.keysym.sym ==SDLK_a){offset_x-=100;}
                    if (event.key.keysym.sym ==SDLK_d){offset_x+=100;}
                    break;
            }
        }
    }
    physicsThread.join();

    return 0;
}
